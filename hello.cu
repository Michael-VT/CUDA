/*
Print Hello World
also print the block id and thread id within the block
*/

#include <hip/hip_runtime.h>
#include <stdio.h> 

const int Nthread = 3; 
const int Nblock = 2; 
 
__global__ void hello(void){
    printf("Hello world! block ID %d, thread ID %d\n",blockIdx.x,threadIdx.x);

}

int main() {
    hello<<<Nblock,Nthread>>>();

}