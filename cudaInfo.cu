// FileNmae: cudaInfo.cu
#include <stdio.h>

#include <hip/hip_runtime_api.h>

int main() 
{
  int deviceCount;
  hipDeviceProp_t deviceProp;

  //������� ��������� CUDA ����������� �� PC.
  hipGetDeviceCount(&deviceCount);

  printf("Device count: %d\n\n", deviceCount);

  for (int i = 0; i < deviceCount; i++)
  {
    //�������� ���������� �� ����������
    hipGetDeviceProperties(&deviceProp, i);

    //������� ��������� �� ����������
    printf("Device name: %s\n", deviceProp.name);
    printf("Total global memory: %zd\n", deviceProp.totalGlobalMem);
    printf("Shared memory per block: %zd\n", deviceProp.sharedMemPerBlock);
    printf("Registers per block: %d\n", deviceProp.regsPerBlock);
    printf("Warp size: %d\n", deviceProp.warpSize);
    printf("Memory pitch: %zd\n", deviceProp.memPitch);
    printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    
    printf("Max threads dimensions: x = %d, y = %d, z = %d\n",
      deviceProp.maxThreadsDim[0],
      deviceProp.maxThreadsDim[1],
      deviceProp.maxThreadsDim[2]);
    
    printf("Max grid size: x = %d, y = %d, z = %d\n", 
      deviceProp.maxGridSize[0], 
      deviceProp.maxGridSize[1], 
      deviceProp.maxGridSize[2]); 

    printf("Clock rate: %d\n", deviceProp.clockRate);
    printf("Total constant memory: %zd\n", deviceProp.totalConstMem); 
    printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Texture alignment: %zd\n", deviceProp.textureAlignment);
    printf("Device overlap: %d\n", deviceProp.deviceOverlap);
    printf("Multiprocessor count: %d\n", deviceProp.multiProcessorCount);

    printf("Kernel execution timeout enabled: %s\n",
      deviceProp.kernelExecTimeoutEnabled ? "true" : "false");
  }

  return 0;
}
